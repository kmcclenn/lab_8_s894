#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <functional>
#include <iostream>
#include <random>
#include <string>
#include <vector>

////////////////////////////////////////////////////////////////////////////////
// Utility Functions

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

class GpuMemoryPool {
  public:
    GpuMemoryPool() = default;

    ~GpuMemoryPool();

    GpuMemoryPool(GpuMemoryPool const &) = delete;
    GpuMemoryPool &operator=(GpuMemoryPool const &) = delete;
    GpuMemoryPool(GpuMemoryPool &&) = delete;
    GpuMemoryPool &operator=(GpuMemoryPool &&) = delete;

    void *alloc(size_t size);
    void reset();

  private:
    std::vector<void *> allocations_;
    std::vector<size_t> capacities_;
    size_t next_idx_ = 0;
};

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

void render_cpu(
    int32_t width,
    int32_t height,
    int32_t n_circle,
    float const *circle_x,
    float const *circle_y,
    float const *circle_radius,
    float const *circle_red,
    float const *circle_green,
    float const *circle_blue,
    float const *circle_alpha,
    float *img_red,
    float *img_green,
    float *img_blue) {

    // Initialize background to white
    for (int32_t pixel_idx = 0; pixel_idx < width * height; pixel_idx++) {
        img_red[pixel_idx] = 1.0f;
        img_green[pixel_idx] = 1.0f;
        img_blue[pixel_idx] = 1.0f;
    }

    // Render circles
    for (int32_t i = 0; i < n_circle; i++) {
        float c_x = circle_x[i];
        float c_y = circle_y[i];
        float c_radius = circle_radius[i];
        for (int32_t y = int32_t(c_y - c_radius); y <= int32_t(c_y + c_radius + 1.0f);
             y++) {
            for (int32_t x = int32_t(c_x - c_radius); x <= int32_t(c_x + c_radius + 1.0f);
                 x++) {
                float dx = x - c_x;
                float dy = y - c_y;
                if (!(0 <= x && x < width && 0 <= y && y < height &&
                      dx * dx + dy * dy < c_radius * c_radius)) {
                    continue;
                }
                int32_t pixel_idx = y * width + x;
                float pixel_red = img_red[pixel_idx];
                float pixel_green = img_green[pixel_idx];
                float pixel_blue = img_blue[pixel_idx];
                float pixel_alpha = circle_alpha[i];
                pixel_red =
                    circle_red[i] * pixel_alpha + pixel_red * (1.0f - pixel_alpha);
                pixel_green =
                    circle_green[i] * pixel_alpha + pixel_green * (1.0f - pixel_alpha);
                pixel_blue =
                    circle_blue[i] * pixel_alpha + pixel_blue * (1.0f - pixel_alpha);
                img_red[pixel_idx] = pixel_red;
                img_green[pixel_idx] = pixel_green;
                img_blue[pixel_idx] = pixel_blue;
            }
        }
    }
}

/// <--- your code here --->

// PSeudo Code

////////////////////////////////////////////////////////////////////////////////
// Optimized GPU Implementation

namespace circles_gpu {

#define THREADS_SCAN (4 * 32)
#define CEIL_DIV(x, y) (((x) + (y) - 1) / (y))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define PAD 32
#define SHMEM_PADDING(idx) ((idx) + ((idx) / PAD))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

/* TODO: your GPU kernels here... */

// __global__ void reduce_uint8(size_t n, uint8_t const *x, uint32_t *out) {

//     size_t base = (size_t)blockIdx.x * (size_t)blockDim.x;
//     size_t offset = base + (size_t)threadIdx.x;

//     extern __shared__ __align__(16) char shmem_raw[];
//     uint32_t *shmem = reinterpret_cast<uint32_t *>(shmem_raw);

//     if (offset < n) {

//         shmem[SHMEM_PADDING(threadIdx.x)] = (uint32_t)(x[offset]);
//     }
//     //  else {
//     //     shmem[SHMEM_PADDING(threadIdx.x)] = 0;
//     // }

//     __syncthreads();

//     for (int i = 1; i < THREADS_SCAN; i <<= 1) {
//         uint32_t add = (threadIdx.x >= i) ? shmem[SHMEM_PADDING(threadIdx.x - i)] : 0;
//         __syncthreads();
//         shmem[SHMEM_PADDING(threadIdx.x)] = add + shmem[SHMEM_PADDING(threadIdx.x)];
//         __syncthreads();
//     }

//     int last = MIN((int)n - (int)(blockIdx.x * blockDim.x), (int)blockDim.x) - 1;
//     out[blockIdx.x] = shmem[SHMEM_PADDING(last)];
// }
__global__ void
reduce_uint8(size_t n, const uint8_t *__restrict__ x, uint32_t *__restrict__ out) {
    const unsigned FULL_MASK = 0xFFFFFFFFu;
    const int WARP_SIZE = 32;

    const size_t base = size_t(blockIdx.x) * blockDim.x;
    const size_t idx = base + threadIdx.x;

    // Load one byte, widen to u32; zero if tail thread is OOB
    uint32_t v = (idx < n) ? (uint32_t)x[idx] : 0u;

    // Intra-warp tree reduction using shfl_down
    int lane = threadIdx.x & (WARP_SIZE - 1);
    int warp = threadIdx.x >> 5; // 0..3 for 128 threads

    for (int off = WARP_SIZE >> 1; off > 0; off >>= 1) {
        v += __shfl_down_sync(FULL_MASK, v, off);
    }

    // Lane 0 of each warp writes its partial sum
    __shared__ uint32_t warp_sums[4]; // 4 warps @ 128 threads
    if (lane == 0)
        warp_sums[warp] = v;
    __syncthreads();

    // Warp 0 reduces the 4 warp sums
    if (warp == 0) {
        uint32_t wv = (lane < 4) ? warp_sums[lane] : 0u;
        // Only two steps needed for 4 lanes (safe to run standard loop too)
        wv += __shfl_down_sync(FULL_MASK, wv, 2);
        wv += __shfl_down_sync(FULL_MASK, wv, 1);
        if (lane == 0)
            out[blockIdx.x] = wv;
    }
}

__global__ void
reduce(size_t n, const uint32_t *__restrict__ x, uint32_t *__restrict__ out) {
    const unsigned FULL_MASK = 0xFFFFFFFFu;
    const int WARP_SIZE = 32;

    const size_t base = size_t(blockIdx.x) * blockDim.x;
    const size_t idx = base + threadIdx.x;

    // Load (zero for out-of-range tail threads)
    uint32_t v = (idx < n) ? x[idx] : 0;

    // Intra-warp reduction (inclusive → final value lands in lane 0)
    int lane = threadIdx.x & (WARP_SIZE - 1);
    int warp = threadIdx.x >> 5; // 0..3 for blockDim=128

    // Tree reduction with shfl_down
    for (int offset = WARP_SIZE >> 1; offset > 0; offset >>= 1) {
        v += __shfl_down_sync(FULL_MASK, v, offset);
    }

    // Each warp writes its sum to shared (lane 0 only)
    __shared__ uint32_t warp_sums[4]; // 4 warps for 128 threads
    if (lane == 0)
        warp_sums[warp] = v;
    __syncthreads();

    // Warp 0 reduces the 4 warp sums using shuffles
    if (warp == 0) {
        // Lanes 0..3 hold the 4 partial sums; others use 0
        uint32_t wv = (lane < 4) ? warp_sums[lane] : 0;

        // Reduce across these 4 lanes (2,1 steps are enough; extra steps add zeros
        // safely)
        wv += __shfl_down_sync(FULL_MASK, wv, 2);
        wv += __shfl_down_sync(FULL_MASK, wv, 1);

        if (lane == 0)
            out[blockIdx.x] = wv;
    }
}

__global__ void scan_block(size_t n, uint32_t const *x, uint32_t *out) {

    extern __shared__ __align__(16) char shmem_raw[];
    uint32_t *shmem = reinterpret_cast<uint32_t *>(shmem_raw);

    int WARP_SIZE = 32;
    const int tid = threadIdx.x;
    const int lane = tid & 31;
    const int warp = tid >> 5;

    uint32_t val = x[tid];

    for (int i = 1; i < WARP_SIZE; i <<= 1) {
        uint32_t v = __shfl_up_sync(0xFFFFFFFFu, val, i);
        if (lane >= i)
            val += v;
    }

    if (lane == WARP_SIZE - 1) {
        shmem[warp] = val;
    }

    __syncthreads();

    if (warp == 0) {
        uint32_t val2 = shmem[lane];

        for (int i = 1; i < WARP_SIZE; i <<= 1) {
            uint32_t v = __shfl_up_sync(0xffffffff, val2, i);
            if (lane >= i)
                val2 += v;
        }

        shmem[lane] = val2;
    }

    __syncthreads();

    uint32_t to_add = (warp == 0) ? 0 : shmem[warp - 1];
    val += to_add;

    out[tid] = val;
}

__global__ void
scan(size_t n, uint32_t const *x, uint32_t const *end_points, uint32_t *out) {

    extern __shared__ __align__(16) char shmem_raw[];
    uint32_t *shmem = reinterpret_cast<uint32_t *>(shmem_raw);

    size_t base = (size_t)blockIdx.x * (size_t)blockDim.x;
    size_t offset = base + (size_t)threadIdx.x;

    if (offset < n) {
        shmem[SHMEM_PADDING(threadIdx.x)] = x[offset];
    }

    __syncthreads();

    for (int i = 1; i < THREADS_SCAN; i <<= 1) {
        uint32_t add = (threadIdx.x >= i) ? shmem[SHMEM_PADDING(threadIdx.x - i)] : 0;
        __syncthreads();
        shmem[SHMEM_PADDING(threadIdx.x)] = add + shmem[SHMEM_PADDING(threadIdx.x)];
        __syncthreads();
    }
    uint32_t block_carry = (blockIdx.x == 0) ? 0 : end_points[blockIdx.x - 1];
    if (offset < n)
        out[offset] = block_carry + shmem[SHMEM_PADDING(threadIdx.x)];
}
// __global__ void scan(
//     size_t n,
//     const uint32_t *__restrict__ x,
//     const uint32_t *__restrict__ end_points, // scanned block totals
//     uint32_t *__restrict__ out) {
//     const unsigned FULL_MASK = 0xFFFFFFFFu;
//     const int WARP_SIZE = 32;

//     const size_t base = size_t(blockIdx.x) * blockDim.x;
//     const size_t idx = base + threadIdx.x;

//     // Load element (0 for OOB lanes so scans stay correct on the tail block)
//     uint32_t v = (idx < n) ? x[idx] : 0u;

//     // Intra-warp inclusive scan
//     const int lane = threadIdx.x & (WARP_SIZE - 1);
//     const int warp = threadIdx.x >> 5; // 0..3 for 128 threads

//     // Mask of active lanes in this warp (those with idx < n)
//     unsigned active = __ballot_sync(FULL_MASK, idx < n);

// #pragma unroll
//     for (int d = 1; d < WARP_SIZE; d <<= 1) {
//         uint32_t u = __shfl_up_sync(active, v, d);
//         if (lane >= d)
//             v += u;
//     }

//     // Write each warp's total to shared; for a partial warp, use its last active lane.
//     __shared__ uint32_t warp_sums[32]; // enough for up to 1024 threads
//     if (active) {
//         // Find last active lane in this warp
//         int last = 31 - __clz(active);
//         if (lane == last)
//             warp_sums[warp] = v;
//     } else {
//         // No active lanes in this warp (happens only on tail blocks beyond n)
//         if (lane == 0)
//             warp_sums[warp] = 0u;
//     }
//     __syncthreads();

//     // Warp 0 scans the warp_sums to get per-warp offsets (inclusive)
//     uint32_t warp_off = 0u;
//     if (warp == 0) {
//         const int WARPS = blockDim.x / WARP_SIZE; // 4 for 128
//         uint32_t t = (lane < WARPS) ? warp_sums[lane] : 0u;

// #pragma unroll
//         for (int d = 1; d < WARP_SIZE; d <<= 1) {
//             uint32_t u = __shfl_up_sync(FULL_MASK, t, d);
//             if (lane >= d)
//                 t += u;
//         }
//         warp_sums[lane] = t; // inclusive prefix of warp totals
//     }
//     __syncthreads();

//     if (warp > 0) {
//         // Exclusive offset from prior warps in this block
//         warp_off = warp_sums[warp - 1];
//     }

//     // Add per-warp offset
//     v += warp_off;

//     // Add block carry from previous blocks (end_points is inclusive scan of block
//     totals) uint32_t block_carry = (blockIdx.x == 0) ? 0u : end_points[blockIdx.x - 1];
//     v += block_carry;

//     // Store result if in range
//     if (idx < n)
//         out[idx] = v;
// }

// Returns desired size of scratch buffer in bytes.
size_t get_workspace_size_scan(size_t n) {
    /* TODO: your CPU code here... */
    size_t total = n;
    size_t size = n;
    while (size > THREADS_SCAN) {
        total += size;
        size = CEIL_DIV(size, THREADS_SCAN);
    }

    return 2 * total * sizeof(uint32_t);
}

__global__ void fill_data(uint8_t *data_uint8, uint32_t *data, uint32_t n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        data[idx] = static_cast<uint32_t>(data_uint8[idx]);
}

// 'launch_scan'
//

// uint32_t *launch_scan(
//     size_t n,
//     uint8_t *x,     // pointer to GPU memory
//     void *workspace // pointer to GPU memory
//     // uint32_t *orig_x // 32 bit representation
// ) {
//     /* TODO: your CPU code here... */
//     uint32_t *arr = reinterpret_cast<uint32_t *>(workspace); // size n
//     uint8_t *data_uint8 = x;
//     uint32_t *data;
//     // uint32_t *orig_x;
//     // compute sums per block

//     size_t size = n;
//     size_t offsets[8];
//     offsets[0] = 0;
//     int iter = 1;

//     // first iteration with uint8
//     fill_data<<<CEIL_DIV(size, THREADS_SCAN), THREADS_SCAN>>>(data_uint8, arr, size);
//     data = arr;
//     offsets[0] = size;

//     if (size > THREADS_SCAN) {
//         size_t blocks = CEIL_DIV(size, THREADS_SCAN);
//         // printf("Reducing with %d blocks and %d threads\n", blocks, THREADS_SCAN);
//         reduce_uint8<<<blocks, THREADS_SCAN, (THREADS_SCAN + PAD) *
//         sizeof(uint32_t)>>>(
//             size,
//             data_uint8,
//             arr + offsets[iter - 1]);

//         size = blocks;
//         data = arr + offsets[iter - 1];
//         offsets[1] = size + offsets[0];
//         iter++;
//     }

//     while (size > THREADS_SCAN) {
//         size_t blocks = CEIL_DIV(size, THREADS_SCAN);
//         reduce<<<blocks, THREADS_SCAN, (THREADS_SCAN + PAD) * sizeof(uint32_t)>>>(
//             size,
//             data,
//             arr + offsets[iter - 1]);

//         size = blocks;
//         data = arr + offsets[iter - 1];

//         offsets[iter] = offsets[iter - 1] + size;
//         iter++;
//     }
//     iter--;

//     uint32_t *final_block = (iter == 0) ? arr : arr + offsets[iter];
//     size_t threads = MIN(THREADS_SCAN, n);

//     uint32_t *base_out = (iter == 0) ? arr : final_block + threads;

//     scan_block<<<1, threads, (32) * sizeof(uint32_t)>>>(size, final_block, base_out);

//     size_t larger_size = threads;
//     base_out += larger_size;
//     uint32_t *end_points = arr + offsets[iter];
//     if (n > THREADS_SCAN) {
//         while (iter >= 0) {

//             larger_size = (iter == 0) ? n : (offsets[iter] - offsets[iter - 1]);
//             size_t blocks = CEIL_DIV(larger_size, THREADS_SCAN);

//             uint32_t *data_ptr =
//                 (iter == 0) ? arr : arr + offsets[iter - 1]; // think this is buggy
//             scan<<<blocks, THREADS_SCAN, (THREADS_SCAN + PAD) * sizeof(uint32_t)>>>(
//                 larger_size,
//                 data_ptr,
//                 end_points,
//                 base_out);

//             size = larger_size;
//             iter--;
//             end_points = base_out;

//             base_out += larger_size;
//         }
//     }
//     base_out -= larger_size;

//     return base_out;
// }

// ============================================================================
// Pass 0: Vectorized widen (uint8_t -> uint32_t) with uchar4 loads
// ============================================================================
__global__ void widen_u8_to_u32_vec4_safe(
    const uint8_t *__restrict__ in,
    uint32_t *__restrict__ out,
    size_t n) {
    size_t i4 = (size_t(blockIdx.x) * blockDim.x + threadIdx.x) * 4;

    // Fast path if base pointer is 4B-aligned
    uintptr_t base = reinterpret_cast<uintptr_t>(in);
    bool aligned4 = (base & 3u) == 0;

    if (i4 + 3 < n) {
        if (aligned4) {
            // Safe to use vector load
            uchar4 v = *reinterpret_cast<const uchar4 *>(in + i4);
            out[i4 + 0] = uint32_t(v.x);
            out[i4 + 1] = uint32_t(v.y);
            out[i4 + 2] = uint32_t(v.z);
            out[i4 + 3] = uint32_t(v.w);
        } else {
            // Fall back to 4 scalar byte loads (still coalesced)
            const uint8_t *p = in + i4;
            out[i4 + 0] = uint32_t(p[0]);
            out[i4 + 1] = uint32_t(p[1]);
            out[i4 + 2] = uint32_t(p[2]);
            out[i4 + 3] = uint32_t(p[3]);
        }
    } else {
        // Tail
        for (int k = 0; k < 4 && i4 + k < n; ++k)
            out[i4 + k] = uint32_t(in[i4 + k]);
    }
}

// ============================================================================
// Pass A: Per-block inclusive scan using shuffles, also writes per-block totals
// blockDim.x must be a multiple of 32; here we assume 128 (4 warps).
// ============================================================================
template <int THREADS>
__global__ void scan_block_write_totals(
    size_t n,
    const uint32_t *__restrict__ x,
    uint32_t *__restrict__ out,
    uint32_t *__restrict__ block_sums) {
    static_assert(THREADS % 32 == 0, "THREADS must be a multiple of 32");
    static_assert(THREADS <= 1024, "THREADS must be <= 1024");

    const unsigned FULL_MASK = 0xFFFFFFFFu;
    const int WARP_SIZE = 32;

    const size_t idx = size_t(blockIdx.x) * blockDim.x + threadIdx.x;
    uint32_t v = (idx < n) ? x[idx] : 0u;

    const int lane = threadIdx.x & (WARP_SIZE - 1);
    const int warp = threadIdx.x >> 5; // warp id within block

// Intra-warp inclusive scan via shuffles
#pragma unroll
    for (int d = 1; d < WARP_SIZE; d <<= 1) {
        uint32_t u = __shfl_up_sync(FULL_MASK, v, d);
        if (lane >= d)
            v += u;
    }

    // Accumulate warp sums in shared
    __shared__ uint32_t warp_sums[THREADS / WARP_SIZE];
    if (lane == WARP_SIZE - 1)
        warp_sums[warp] = v;
    __syncthreads();

    // Warp 0 scans warp_sums
    uint32_t warp_off = 0;
    if (warp == 0) {
        const int WARPS = THREADS / WARP_SIZE;
        uint32_t t = (lane < WARPS) ? warp_sums[lane] : 0u;
#pragma unroll
        for (int d = 1; d < WARP_SIZE; d <<= 1) {
            uint32_t u = __shfl_up_sync(FULL_MASK, t, d);
            if (lane >= d)
                t += u;
        }
        warp_sums[lane] = t;
    }
    __syncthreads();

    if (warp > 0)
        warp_off = warp_sums[warp - 1]; // what to sum
    v += warp_off;

    if (idx < n)
        out[idx] = v;
    // scan per block

    // Last active thread in the block writes the block's total
    if (threadIdx.x == blockDim.x - 1 || idx == n - 1) {
        block_sums[blockIdx.x] = v;
    }
}

// ============================================================================
// Pass B (small-array scan): Single-CTA inclusive scan for up to 1024 elems
// Launch with T threads (multiple of 32, T <= 1024), dynamic smem = (#warps)*4
// ============================================================================
__global__ void scan_singleCTA_inclusive(
    size_t m,
    const uint32_t *__restrict__ in,
    uint32_t *__restrict__ out) {
    extern __shared__ uint32_t smem[]; // holds warp sums (WARPS elements)
    const unsigned FULL_MASK = 0xFFFFFFFFu;
    const int WARP_SIZE = 32;

    const int tid = threadIdx.x;
    const int lane = tid & (WARP_SIZE - 1);
    const int warp = tid >> 5;
    const int WARPS = (blockDim.x + 31) / 32;

    uint32_t v = (size_t(tid) < m) ? in[tid] : 0u;

// Intra-warp inclusive
#pragma unroll
    for (int d = 1; d < WARP_SIZE; d <<= 1) {
        uint32_t u = __shfl_up_sync(FULL_MASK, v, d);
        if (lane >= d)
            v += u;
    }

    if (lane == WARP_SIZE - 1)
        smem[warp] = v;
    __syncthreads();

    uint32_t warp_off = 0;
    if (warp == 0) {
        uint32_t t = (lane < WARPS) ? smem[lane] : 0u;
#pragma unroll
        for (int d = 1; d < WARP_SIZE; d <<= 1) {
            uint32_t u = __shfl_up_sync(FULL_MASK, t, d);
            if (lane >= d)
                t += u;
        }
        smem[lane] = t;
    }
    __syncthreads();

    if (warp > 0)
        warp_off = smem[warp - 1];
    v += warp_off;

    if (size_t(tid) < m)
        out[tid] = v;
}

// ============================================================================
// Pass C: Uniform add scanned block-prefix to each element
// ============================================================================
__global__ void uniform_add(
    size_t n,
    uint32_t *__restrict__ out,
    const uint32_t *__restrict__ block_prefix) {
    const size_t idx = size_t(blockIdx.x) * blockDim.x + threadIdx.x;
    if (idx >= n)
        return;
    const uint32_t add = (blockIdx.x == 0) ? 0u : block_prefix[blockIdx.x - 1];
    out[idx] += add;
}

// ============================================================================
// Workspace sizing helper for the 3-pass scan
// Layout: [n] u32_out | [blocksA] block_sums | [blocksA] prefix_L0 |
//         [blocksB0] tmp0 | [blocksB0] tmp1
// ============================================================================
inline size_t get_workspace_size_scan_fast(size_t n, int threads = 128) {
    const size_t blocksA = CEIL_DIV(n, (size_t)threads);
    const size_t blocksB0 = CEIL_DIV(blocksA, (size_t)threads);
    const size_t u32_elems = n + blocksA + blocksA + blocksB0 + blocksB0;
    return u32_elems * sizeof(uint32_t);
}

// ============================================================================
// Main API: 3-pass inclusive scan from uint8_t (d_in) to uint32_t (result)
// Returns a device pointer (inside `workspace`) to the n-length u32 scan.
// threads is fixed to 128 in this implementation (4 warps).
// ============================================================================
uint32_t *launch_scan(size_t n, const uint8_t *d_in, void *workspace, int threads = 128) {
    if (n == 0)
        return nullptr;
    if (threads != 128) {
        fprintf(stderr, "launch_scan: this build assumes threads==128.\n");
        std::exit(2);
    }

    auto *base = reinterpret_cast<uint32_t *>(workspace);
    const size_t blocksA = CEIL_DIV(n, (size_t)threads);
    const size_t blocksB0 = CEIL_DIV(blocksA, (size_t)threads);

    uint32_t *d_out_u32 = base;                     // [0, n)
    uint32_t *d_block_sums = d_out_u32 + n;         // [n, n+blocksA)
    uint32_t *d_prefix_L0 = d_block_sums + blocksA; // [.. + blocksA)
    uint32_t *d_tmp0 = d_prefix_L0 + blocksA;       // [.. + blocksB0)
    uint32_t *d_tmp1 = d_tmp0 + blocksB0;           // [.. + blocksB0)

    // --- Pass 0: widen u8 -> u32 (vectorized loads) ---
    {
        const int t = threads;
        const int b = int(CEIL_DIV(n, (size_t)(4 * t)));
        if (b > 0)
            widen_u8_to_u32_vec4_safe<<<b, t>>>(d_in, d_out_u32, n);
    }

    // --- Pass A: per-block inclusive scan + block totals ---
    {
        const int t = threads;
        const int b = int(blocksA);
        if (b > 0)
            scan_block_write_totals<128><<<b, t>>>(n, d_out_u32, d_out_u32, d_block_sums);
    }

    // --- Pass B: scan the per-block totals ---
    if (blocksA <= 1024) {
        const int T = 1024; // threads for single-CTA scan
        const int WARPS = (T + 31) / 32;
        const int smem = WARPS * sizeof(uint32_t);
        scan_singleCTA_inclusive<<<1, T, smem>>>(blocksA, d_block_sums, d_prefix_L0);
    } else {
        // Level 0: scan block_sums in blocks; write prefix_L0; accumulate level-1 sums in
        // d_tmp0
        {
            const int t = threads;
            const int b = int(blocksB0);
            scan_block_write_totals<128>
                <<<b, t>>>(blocksA, d_block_sums, d_prefix_L0, d_tmp0);
        }
        // Level 1: scan the level-1 sums with a single CTA into d_tmp1
        {
            const int T = 1024;
            const int WARPS = (T + 31) / 32;
            const int smem = WARPS * sizeof(uint32_t);
            scan_singleCTA_inclusive<<<1, T, smem>>>(blocksB0, d_tmp0, d_tmp1);
        }
        // Propagate level-1 prefixes back into level-0 prefix
        uniform_add<<<int(blocksB0), threads>>>(blocksA, d_prefix_L0, d_tmp1);
    }

    // --- Pass C: uniform add scanned block prefixes into full output ---
    uniform_add<<<int(blocksA), threads>>>(n, d_out_u32, d_prefix_L0);

    // Optionally synchronize here if the caller expects completion.
    // CUDA_CHECK(cudaDeviceSynchronize());

    return d_out_u32; // result lives in the first n uint32_t of workspace
}

#define THREADS_X 32
#define THREADS_Y 32
#define TILE_SIZE 64

/* TODO: your GPU kernels here... */

__global__ void tile_coverage(
    int32_t width,
    int32_t height,
    int32_t n_circle,
    float const *circle_x,
    float const *circle_y,
    float const *circle_radius,
    uint8_t *circle_map,
    int32_t num_tiles) {

    int32_t circle = blockIdx.x * blockDim.x + threadIdx.x;

    size_t stride = (size_t)n_circle;
    size_t total = (size_t)num_tiles * stride;

    if (circle < n_circle) {

        int tile_width = width / TILE_SIZE;
        int tile_height = height / TILE_SIZE;

        float x = circle_x[circle];
        float y = circle_y[circle];
        float rad = circle_radius[circle];

        int left = (int)MAX((x - rad) / TILE_SIZE, 0);
        int right = (int)MIN((x + rad) / TILE_SIZE, tile_width - 1);
        int top = (int)MAX((y - rad) / TILE_SIZE, 0);
        int bot = (int)MIN((y + rad) / TILE_SIZE, tile_height - 1);

        // go through each corner of the bounding box
        for (int x = left; x < right + 1; x += 1) {
            for (int y = top; y < bot + 1; y += 1) {
                int tile = y * tile_width + x;

                size_t idx = (size_t)tile * stride + (size_t)circle;

                circle_map[idx] = (idx < total);
            }
        }
    }
}
#define MAX_CIRCLES_PER_TILE 44000
__global__ void compact_stream(
    uint32_t n_circle,
    uint32_t *scanned_idxs,
    uint32_t *compacted_idxs,
    uint32_t *num_circles_per_tile) {

    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n_circle) {
        if (idx == 0) {
            compacted_idxs[0] = 0;
        } else if (scanned_idxs[idx] != scanned_idxs[idx - 1]) {
            uint32_t c_idx = scanned_idxs[idx];

            compacted_idxs[c_idx - 1] = idx;
        }
    }
    if (idx == n_circle - 1) {
        *num_circles_per_tile = scanned_idxs[idx];
    }
}

#define CIRCLE_STATS 8
__global__ void render_pixels(
    uint32_t *num_circles_per_tile,
    uint32_t n_circle,
    uint32_t num_tiles,
    int32_t width,
    int32_t height,
    float const *circle_x,
    float const *circle_y,
    float const *circle_radius,
    float const *circle_red,
    float const *circle_green,
    float const *circle_blue,
    float const *circle_alpha,
    float *img_red,
    float *img_green,
    float *img_blue,
    uint32_t *compacted_stream

    // float **x_coal_ptr
) {

    int blockToTile =
        TILE_SIZE / THREADS_X; // assume threads x and threads y are the same

    int32_t tile_idx = blockIdx.y * gridDim.x + blockIdx.x;

    int32_t base_pixel_row = (blockIdx.y * blockDim.y) * blockToTile + threadIdx.y;
    int32_t base_pixel_col = (blockIdx.x * blockDim.x) * blockToTile + threadIdx.x;

    extern __shared__ __align__(16) char shmem_raw[];
    float *shmem = reinterpret_cast<float *>(shmem_raw);
    const int threads = blockDim.x * blockDim.y;

    uint32_t n_t_circles = num_circles_per_tile[tile_idx];
    uint32_t *tile_circles = compacted_stream + tile_idx * MAX_CIRCLES_PER_TILE;

    float red_out[4] = {1.0f, 1.0f, 1.0f, 1.0f};
    float green_out[4] = {1.0f, 1.0f, 1.0f, 1.0f};
    float blue_out[4] = {1.0f, 1.0f, 1.0f, 1.0f};

    for (int chunk = 0; chunk < n_t_circles; chunk += threads) {

        int tid = (threadIdx.y * blockDim.x + threadIdx.x);
        int circle_id_id = chunk + tid;

        if (circle_id_id < n_t_circles) {
            uint32_t c_idx = tile_circles[circle_id_id];

            float4 circle_xyra = make_float4(
                circle_x[c_idx],
                circle_y[c_idx],
                circle_radius[c_idx],
                circle_alpha[c_idx]);

            float4 circle_rgbp = make_float4(
                circle_red[c_idx],
                circle_green[c_idx],
                circle_blue[c_idx],
                0.0f // padding (unused)
            );

            reinterpret_cast<float4 *>(&shmem[CIRCLE_STATS * tid])[0] = circle_xyra;
            reinterpret_cast<float4 *>(&shmem[CIRCLE_STATS * tid])[1] = circle_rgbp;
        }

        int loop_iters = threads;
        if (chunk + threads >= n_t_circles) {
            loop_iters = n_t_circles - chunk; // the remainder
        }

        __syncthreads();
        for (int b_i = 0; b_i < blockToTile; ++b_i) {
            for (int b_j = 0; b_j < blockToTile; ++b_j) {

                int b_id = b_i * blockToTile + b_j;
                int32_t pixel_row = base_pixel_row + b_i * THREADS_Y;
                int32_t pixel_col = base_pixel_col + b_j * THREADS_X;

                for (int i = 0; i < loop_iters; ++i) {

                    // uint32_t c_idx = tile_circles[i];
                    float *circle_stats = shmem + CIRCLE_STATS * i;

                    float4 xyra = reinterpret_cast<const float4 *>(circle_stats)[0];
                    float4 rgbp = reinterpret_cast<const float4 *>(circle_stats)[1];

                    float x = xyra.x;
                    float y = xyra.y;
                    float rad = xyra.z;
                    float alpha = xyra.w;

                    float c_red = rgbp.x;
                    float c_green = rgbp.y;
                    float c_blue = rgbp.z;

                    float dy = pixel_row - y;
                    float dx = pixel_col - x;

                    if (!(dy * dy + dx * dx < rad * rad))
                        continue;

                    red_out[b_id] = c_red * alpha + red_out[b_id] * (1.0f - alpha);
                    green_out[b_id] = c_green * alpha + green_out[b_id] * (1.0f - alpha);
                    blue_out[b_id] = c_blue * alpha + blue_out[b_id] * (1.0f - alpha);
                }
            }
        }
        // __syncthreads();
    }

    for (int i = 0; i < blockToTile; ++i) {
        for (int j = 0; j < blockToTile; ++j) {

            int32_t pixel_row = base_pixel_row + i * THREADS_Y;
            int32_t pixel_col = base_pixel_col + j * THREADS_X;

            int32_t pixel_idx = pixel_row * width + pixel_col;
            int b_id = i * blockToTile + j;

            img_red[pixel_idx] = red_out[b_id];
            img_green[pixel_idx] = green_out[b_id];
            img_blue[pixel_idx] = blue_out[b_id];
        }
    }
} // namespace circles_gpu

void launch_render(
    int32_t width,
    int32_t height,
    int32_t n_circle,
    float const *circle_x,      // pointer to GPU memory
    float const *circle_y,      // pointer to GPU memory
    float const *circle_radius, // pointer to GPU memory
    float const *circle_red,    // pointer to GPU memory
    float const *circle_green,  // pointer to GPU memory
    float const *circle_blue,   // pointer to GPU memory
    float const *circle_alpha,  // pointer to GPU memory
    float *img_red,             // pointer to GPU memory
    float *img_green,           // pointer to GPU memory
    float *img_blue,            // pointer to GPU memory
    GpuMemoryPool &memory_pool) {

    uint32_t threads_c = THREADS_X * THREADS_Y;
    uint32_t blocks_c = CEIL_DIV(n_circle, threads_c);

    // Get circle coverage
    int32_t num_tiles = CEIL_DIV(width, TILE_SIZE) * CEIL_DIV(height, TILE_SIZE);
    size_t circle_map_size = (size_t)num_tiles * (size_t)n_circle * sizeof(uint8_t);

    uint8_t *circle_map = reinterpret_cast<uint8_t *>(memory_pool.alloc(circle_map_size));

    tile_coverage<<<blocks_c, threads_c>>>(
        width,
        height,
        n_circle,
        circle_x,
        circle_y,
        circle_radius,
        circle_map,
        num_tiles);

    size_t scan_size = get_workspace_size_scan(n_circle);

    uint32_t **tile_circle_idxs =
        reinterpret_cast<uint32_t **>(memory_pool.alloc(num_tiles * sizeof(uint32_t *)));

    uint32_t *num_circles_per_tile = reinterpret_cast<uint32_t *>(
        memory_pool.alloc(num_tiles * sizeof(uint32_t))); //[num_tiles];
    void *scan_workspace = memory_pool.alloc(scan_size);

    // size_t mtemp = 0;

    uint32_t *compacted_stream = reinterpret_cast<uint32_t *>(
        memory_pool.alloc(num_tiles * MAX_CIRCLES_PER_TILE * sizeof(uint32_t)));

    for (int i = 0; i < num_tiles; ++i) {

        uint32_t *scanned_circle_idxs = launch_scan(
            (size_t)n_circle,
            circle_map + (size_t)i * (size_t)n_circle,
            scan_workspace);

        compact_stream<<<blocks_c, threads_c>>>(
            n_circle,
            scanned_circle_idxs,
            compacted_stream + i * MAX_CIRCLES_PER_TILE,
            num_circles_per_tile + i);
    }

    dim3 threads_p = dim3(THREADS_X, THREADS_Y);
    dim3 blocks_p = dim3(CEIL_DIV(width, TILE_SIZE), CEIL_DIV(height, TILE_SIZE));

    size_t render_shmem = sizeof(float) * (CIRCLE_STATS * THREADS_X * THREADS_Y);
    // CUDA_CHECK(cudaFuncSetAttribute(
    //     render_pixels,
    //     cudaFuncAttributeMaxDynamicSharedMemorySize,
    //     render_shmem));

    render_pixels<<<blocks_p, threads_p, render_shmem>>>(
        num_circles_per_tile,
        n_circle,
        num_tiles,
        width,
        height,
        circle_x,
        circle_y,
        circle_radius,
        circle_red,
        circle_green,
        circle_blue,
        circle_alpha,
        img_red,
        img_green,
        img_blue,
        compacted_stream);
}

} // namespace circles_gpu

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

GpuMemoryPool::~GpuMemoryPool() {
    for (auto ptr : allocations_) {
        CUDA_CHECK(hipFree(ptr));
    }
}

void *GpuMemoryPool::alloc(size_t size) {
    if (next_idx_ < allocations_.size()) {
        auto idx = next_idx_++;
        if (size > capacities_.at(idx)) {
            CUDA_CHECK(hipFree(allocations_.at(idx)));
            CUDA_CHECK(hipMalloc(&allocations_.at(idx), size));
            CUDA_CHECK(hipMemset(allocations_.at(idx), 0, size));
            capacities_.at(idx) = size;
        }
        return allocations_.at(idx);
    } else {
        void *ptr;
        CUDA_CHECK(hipMalloc(&ptr, size));
        CUDA_CHECK(hipMemset(ptr, 0, size));
        allocations_.push_back(ptr);
        capacities_.push_back(size);
        next_idx_++;
        return ptr;
    }
}

void GpuMemoryPool::reset() {
    next_idx_ = 0;
    for (int32_t i = 0; i < allocations_.size(); i++) {
        CUDA_CHECK(hipMemset(allocations_.at(i), 0, capacities_.at(i)));
    }
}

template <typename Reset, typename F>
double benchmark_ms(double target_time_ms, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        f();
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms);
    }
    return best_time_ms;
}

struct Scene {
    int32_t width;
    int32_t height;
    std::vector<float> circle_x;
    std::vector<float> circle_y;
    std::vector<float> circle_radius;
    std::vector<float> circle_red;
    std::vector<float> circle_green;
    std::vector<float> circle_blue;
    std::vector<float> circle_alpha;

    int32_t n_circle() const { return circle_x.size(); }
};

struct Image {
    int32_t width;
    int32_t height;
    std::vector<float> red;
    std::vector<float> green;
    std::vector<float> blue;
};

float max_abs_diff(Image const &a, Image const &b) {
    float max_diff = 0.0f;
    for (int32_t idx = 0; idx < a.width * a.height; idx++) {
        float diff_red = std::abs(a.red.at(idx) - b.red.at(idx));
        float diff_green = std::abs(a.green.at(idx) - b.green.at(idx));
        float diff_blue = std::abs(a.blue.at(idx) - b.blue.at(idx));
        max_diff = std::max(max_diff, diff_red);
        max_diff = std::max(max_diff, diff_green);
        max_diff = std::max(max_diff, diff_blue);
    }
    return max_diff;
}

struct Results {
    bool correct;
    float max_abs_diff;
    Image image_expected;
    Image image_actual;
    double time_ms;
};

enum class Mode {
    TEST,
    BENCHMARK,
};

template <typename T> struct GpuBuf {
    T *data;

    explicit GpuBuf(size_t n) { CUDA_CHECK(hipMalloc(&data, n * sizeof(T))); }

    explicit GpuBuf(std::vector<T> const &host_data) {
        CUDA_CHECK(hipMalloc(&data, host_data.size() * sizeof(T)));
        CUDA_CHECK(hipMemcpy(
            data,
            host_data.data(),
            host_data.size() * sizeof(T),
            hipMemcpyHostToDevice));
    }

    ~GpuBuf() { CUDA_CHECK(hipFree(data)); }
};

Results run_config(Mode mode, Scene const &scene) {
    auto img_expected = Image{
        scene.width,
        scene.height,
        std::vector<float>(scene.height * scene.width, 0.0f),
        std::vector<float>(scene.height * scene.width, 0.0f),
        std::vector<float>(scene.height * scene.width, 0.0f)};

    render_cpu(
        scene.width,
        scene.height,
        scene.n_circle(),
        scene.circle_x.data(),
        scene.circle_y.data(),
        scene.circle_radius.data(),
        scene.circle_red.data(),
        scene.circle_green.data(),
        scene.circle_blue.data(),
        scene.circle_alpha.data(),
        img_expected.red.data(),
        img_expected.green.data(),
        img_expected.blue.data());

    auto circle_x_gpu = GpuBuf<float>(scene.circle_x);
    auto circle_y_gpu = GpuBuf<float>(scene.circle_y);
    auto circle_radius_gpu = GpuBuf<float>(scene.circle_radius);
    auto circle_red_gpu = GpuBuf<float>(scene.circle_red);
    auto circle_green_gpu = GpuBuf<float>(scene.circle_green);
    auto circle_blue_gpu = GpuBuf<float>(scene.circle_blue);
    auto circle_alpha_gpu = GpuBuf<float>(scene.circle_alpha);
    auto img_red_gpu = GpuBuf<float>(scene.height * scene.width);
    auto img_green_gpu = GpuBuf<float>(scene.height * scene.width);
    auto img_blue_gpu = GpuBuf<float>(scene.height * scene.width);

    auto memory_pool = GpuMemoryPool();

    auto reset = [&]() {
        CUDA_CHECK(
            hipMemset(img_red_gpu.data, 0, scene.height * scene.width * sizeof(float)));
        CUDA_CHECK(hipMemset(
            img_green_gpu.data,
            0,
            scene.height * scene.width * sizeof(float)));
        CUDA_CHECK(
            hipMemset(img_blue_gpu.data, 0, scene.height * scene.width * sizeof(float)));
        memory_pool.reset();
    };

    auto f = [&]() {
        circles_gpu::launch_render(
            scene.width,
            scene.height,
            scene.n_circle(),
            circle_x_gpu.data,
            circle_y_gpu.data,
            circle_radius_gpu.data,
            circle_red_gpu.data,
            circle_green_gpu.data,
            circle_blue_gpu.data,
            circle_alpha_gpu.data,
            img_red_gpu.data,
            img_green_gpu.data,
            img_blue_gpu.data,
            memory_pool);
    };

    reset();
    f();

    auto img_actual = Image{
        scene.width,
        scene.height,
        std::vector<float>(scene.height * scene.width, 0.0f),
        std::vector<float>(scene.height * scene.width, 0.0f),
        std::vector<float>(scene.height * scene.width, 0.0f)};

    CUDA_CHECK(hipMemcpy(
        img_actual.red.data(),
        img_red_gpu.data,
        scene.height * scene.width * sizeof(float),
        hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(
        img_actual.green.data(),
        img_green_gpu.data,
        scene.height * scene.width * sizeof(float),
        hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(
        img_actual.blue.data(),
        img_blue_gpu.data,
        scene.height * scene.width * sizeof(float),
        hipMemcpyDeviceToHost));

    float max_diff = max_abs_diff(img_expected, img_actual);

    if (max_diff > 5e-2) {
        return Results{
            false,
            max_diff,
            std::move(img_expected),
            std::move(img_actual),
            0.0,
        };
    }

    if (mode == Mode::TEST) {
        return Results{
            true,
            max_diff,
            std::move(img_expected),
            std::move(img_actual),
            0.0,
        };
    }

    double time_ms = benchmark_ms(1000.0, reset, f);

    return Results{
        true,
        max_diff,
        std::move(img_expected),
        std::move(img_actual),
        time_ms,
    };
}

template <typename Rng>
Scene gen_random(Rng &rng, int32_t width, int32_t height, int32_t n_circle) {
    auto unif_0_1 = std::uniform_real_distribution<float>(0.0f, 1.0f);
    auto z_values = std::vector<float>();
    for (int32_t i = 0; i < n_circle; i++) {
        float z;
        for (;;) {
            z = unif_0_1(rng);
            z = std::max(z, unif_0_1(rng));
            if (z > 0.01) {
                break;
            }
        }
        // float z = std::max(unif_0_1(rng), unif_0_1(rng));
        z_values.push_back(z);
    }
    std::sort(z_values.begin(), z_values.end(), std::greater<float>());

    auto colors = std::vector<uint32_t>{
        0xd32360,
        0xcc9f26,
        0x208020,
        0x2874aa,
    };
    auto color_idx_dist = std::uniform_int_distribution<int>(0, colors.size() - 1);
    auto alpha_dist = std::uniform_real_distribution<float>(0.0f, 0.3f);

    int32_t fog_interval = n_circle / 10;
    float fog_alpha = 0.2;

    auto scene = Scene{width, height};
    float base_radius_scale = 1.0f;
    int32_t i = 0;
    for (float z : z_values) {
        float max_radius = base_radius_scale / z;
        float radius = std::max(1.0f, unif_0_1(rng) * max_radius);
        float x = unif_0_1(rng) * (width + 2 * max_radius) - max_radius;
        float y = unif_0_1(rng) * (height + 2 * max_radius) - max_radius;
        int color_idx = color_idx_dist(rng);
        uint32_t color = colors[color_idx];
        scene.circle_x.push_back(x);
        scene.circle_y.push_back(y);
        scene.circle_radius.push_back(radius);
        scene.circle_red.push_back(float((color >> 16) & 0xff) / 255.0f);
        scene.circle_green.push_back(float((color >> 8) & 0xff) / 255.0f);
        scene.circle_blue.push_back(float(color & 0xff) / 255.0f);
        scene.circle_alpha.push_back(alpha_dist(rng));
        i++;
        if (i % fog_interval == 0 && i + 1 < n_circle) {
            scene.circle_x.push_back(float(width - 1) / 2.0f);
            scene.circle_y.push_back(float(height - 1) / 2.0f);
            scene.circle_radius.push_back(float(std::max(width, height)));
            scene.circle_red.push_back(1.0f);
            scene.circle_green.push_back(1.0f);
            scene.circle_blue.push_back(1.0f);
            scene.circle_alpha.push_back(fog_alpha);
        }
    }

    return scene;
}

constexpr float PI = 3.14159265359f;

Scene gen_overlapping_opaque() {
    int32_t width = 256;
    int32_t height = 256;

    auto scene = Scene{width, height};

    auto colors = std::vector<uint32_t>{
        0xd32360,
        0xcc9f26,
        0x208020,
        0x2874aa,
    };

    int32_t n_circle = 20;
    int32_t n_ring = 4;
    float angle_range = PI;
    for (int32_t ring = 0; ring < n_ring; ring++) {
        float dist = 20.0f * (ring + 1);
        float saturation = float(ring + 1) / n_ring;
        float hue_shift = float(ring) / (n_ring - 1);
        for (int32_t i = 0; i < n_circle; i++) {
            float theta = angle_range * i / (n_circle - 1);
            float x = width / 2.0f - dist * std::cos(theta);
            float y = height / 2.0f - dist * std::sin(theta);
            scene.circle_x.push_back(x);
            scene.circle_y.push_back(y);
            scene.circle_radius.push_back(16.0f);
            auto color = colors[(i + ring * 2) % colors.size()];
            scene.circle_red.push_back(float((color >> 16) & 0xff) / 255.0f);
            scene.circle_green.push_back(float((color >> 8) & 0xff) / 255.0f);
            scene.circle_blue.push_back(float(color & 0xff) / 255.0f);
            scene.circle_alpha.push_back(1.0f);
        }
    }

    return scene;
}

Scene gen_overlapping_transparent() {
    int32_t width = 256;
    int32_t height = 256;

    auto scene = Scene{width, height};

    float offset = 20.0f;
    float radius = 40.0f;
    scene.circle_x = std::vector<float>{
        (width - 1) / 2.0f - offset,
        (width - 1) / 2.0f + offset,
        (width - 1) / 2.0f + offset,
        (width - 1) / 2.0f - offset,
    };
    scene.circle_y = std::vector<float>{
        (height - 1) * 0.75f,
        (height - 1) * 0.75f,
        (height - 1) * 0.25f,
        (height - 1) * 0.25f,
    };
    scene.circle_radius = std::vector<float>{
        radius,
        radius,
        radius,
        radius,
    };
    // 0xd32360
    // 0x2874aa
    scene.circle_red = std::vector<float>{
        float(0xd3) / 255.0f,
        float(0x28) / 255.0f,
        float(0x28) / 255.0f,
        float(0xd3) / 255.0f,
    };
    scene.circle_green = std::vector<float>{
        float(0x23) / 255.0f,
        float(0x74) / 255.0f,
        float(0x74) / 255.0f,
        float(0x23) / 255.0f,
    };
    scene.circle_blue = std::vector<float>{
        float(0x60) / 255.0f,
        float(0xaa) / 255.0f,
        float(0xaa) / 255.0f,
        float(0x60) / 255.0f,
    };
    scene.circle_alpha = std::vector<float>{
        0.75f,
        0.75f,
        0.75f,
        0.75f,
    };
    return scene;
}

Scene gen_simple() {
    /*
        0xd32360,
        0xcc9f26,
        0x208020,
        0x2874aa,
    */
    int32_t width = 256;
    int32_t height = 256;
    auto scene = Scene{width, height};
    scene.circle_x = std::vector<float>{
        (width - 1) * 0.25f,
        (width - 1) * 0.75f,
        (width - 1) * 0.25f,
        (width - 1) * 0.75f,
    };
    scene.circle_y = std::vector<float>{
        (height - 1) * 0.25f,
        (height - 1) * 0.25f,
        (height - 1) * 0.75f,
        (height - 1) * 0.75f,
    };
    scene.circle_radius = std::vector<float>{
        40.0f,
        40.0f,
        40.0f,
        40.0f,
    };
    scene.circle_red = std::vector<float>{
        float(0xd3) / 255.0f,
        float(0xcc) / 255.0f,
        float(0x20) / 255.0f,
        float(0x28) / 255.0f,
    };
    scene.circle_green = std::vector<float>{
        float(0x23) / 255.0f,
        float(0x9f) / 255.0f,
        float(0x80) / 255.0f,
        float(0x74) / 255.0f,
    };
    scene.circle_blue = std::vector<float>{
        float(0x60) / 255.0f,
        float(0x26) / 255.0f,
        float(0x20) / 255.0f,
        float(0xaa) / 255.0f,
    };
    scene.circle_alpha = std::vector<float>{
        1.0f,
        1.0f,
        1.0f,
        1.0f,
    };
    return scene;
}

// Output image writers: BMP file header structure
#pragma pack(push, 1)
struct BMPHeader {
    uint16_t fileType{0x4D42};   // File type, always "BM"
    uint32_t fileSize{0};        // Size of the file in bytes
    uint16_t reserved1{0};       // Always 0
    uint16_t reserved2{0};       // Always 0
    uint32_t dataOffset{54};     // Start position of pixel data
    uint32_t headerSize{40};     // Size of this header (40 bytes)
    int32_t width{0};            // Image width in pixels
    int32_t height{0};           // Image height in pixels
    uint16_t planes{1};          // Number of color planes
    uint16_t bitsPerPixel{24};   // Bits per pixel (24 for RGB)
    uint32_t compression{0};     // Compression method (0 for uncompressed)
    uint32_t imageSize{0};       // Size of raw bitmap data
    int32_t xPixelsPerMeter{0};  // Horizontal resolution
    int32_t yPixelsPerMeter{0};  // Vertical resolution
    uint32_t colorsUsed{0};      // Number of colors in the color palette
    uint32_t importantColors{0}; // Number of important colors
};
#pragma pack(pop)

void write_bmp(
    std::string const &fname,
    uint32_t width,
    uint32_t height,
    const std::vector<uint8_t> &pixels) {
    BMPHeader header;
    header.width = width;
    header.height = height;

    uint32_t rowSize = (width * 3 + 3) & (~3); // Align to 4 bytes
    header.imageSize = rowSize * height;
    header.fileSize = header.dataOffset + header.imageSize;

    std::ofstream file(fname, std::ios::binary);
    file.write(reinterpret_cast<const char *>(&header), sizeof(header));

    // Write pixel data with padding
    std::vector<uint8_t> padding(rowSize - width * 3, 0);
    for (int32_t idx_y = height - 1; idx_y >= 0;
         --idx_y) { // BMP stores pixels from bottom to top
        const uint8_t *row = &pixels[idx_y * width * 3];
        file.write(reinterpret_cast<const char *>(row), width * 3);
        if (!padding.empty()) {
            file.write(reinterpret_cast<const char *>(padding.data()), padding.size());
        }
    }
}

uint8_t float_to_byte(float x) {
    if (x < 0) {
        return 0;
    } else if (x >= 1) {
        return 255;
    } else {
        return x * 255.0f;
    }
}

void write_image(std::string const &fname, Image const &img) {
    auto pixels = std::vector<uint8_t>(img.width * img.height * 3);
    for (int32_t idx = 0; idx < img.width * img.height; idx++) {
        float red = img.red.at(idx);
        float green = img.green.at(idx);
        float blue = img.blue.at(idx);
        // BMP stores pixels in BGR order
        pixels.at(idx * 3) = float_to_byte(blue);
        pixels.at(idx * 3 + 1) = float_to_byte(green);
        pixels.at(idx * 3 + 2) = float_to_byte(red);
    }
    write_bmp(fname, img.width, img.height, pixels);
}

Image compute_img_diff(Image const &a, Image const &b) {
    auto img_diff = Image{
        a.width,
        a.height,
        std::vector<float>(a.height * a.width, 0.0f),
        std::vector<float>(a.height * a.width, 0.0f),
        std::vector<float>(a.height * a.width, 0.0f),
    };
    for (int32_t idx = 0; idx < a.width * a.height; idx++) {
        img_diff.red.at(idx) = std::abs(a.red.at(idx) - b.red.at(idx));
        img_diff.green.at(idx) = std::abs(a.green.at(idx) - b.green.at(idx));
        img_diff.blue.at(idx) = std::abs(a.blue.at(idx) - b.blue.at(idx));
    }
    return img_diff;
}

struct SceneTest {
    std::string name;
    Mode mode;
    Scene scene;
};

int main(int argc, char const *const *argv) {
    auto rng = std::mt19937(0xCA7CAFE);

    auto scenes = std::vector<SceneTest>();
    scenes.push_back({"simple", Mode::TEST, gen_simple()});
    scenes.push_back({"overlapping_opaque", Mode::TEST, gen_overlapping_opaque()});
    scenes.push_back(
        {"overlapping_transparent", Mode::TEST, gen_overlapping_transparent()});
    scenes.push_back(
        {"ten_million_circles",
         Mode::BENCHMARK,
         gen_random(rng, 1024, 1024, 10'000'000)});

    int32_t fail_count = 0;

    int32_t count = 0;
    for (auto const &scene_test : scenes) {
        auto i = count++;
        printf("\nTesting scene '%s'\n", scene_test.name.c_str());
        auto results = run_config(scene_test.mode, scene_test.scene);
        write_image(
            std::string("out/img") + std::to_string(i) + "_" + scene_test.name +
                "_cpu.bmp",
            results.image_expected);
        write_image(
            std::string("out/img") + std::to_string(i) + "_" + scene_test.name +
                "_gpu.bmp",
            results.image_actual);
        if (!results.correct) {
            printf("  Result did not match expected image\n");
            printf("  Max absolute difference: %.2e\n", results.max_abs_diff);
            auto diff = compute_img_diff(results.image_expected, results.image_actual);
            write_image(
                std::string("out/img") + std::to_string(i) + "_" + scene_test.name +
                    "_diff.bmp",
                diff);
            printf(
                "  (Wrote image diff to 'out/img%d_%s_diff.bmp')\n",
                i,
                scene_test.name.c_str());
            fail_count++;
            continue;
        } else {
            printf("  OK\n");
        }
        if (scene_test.mode == Mode::BENCHMARK) {
            printf("  Time: %f ms\n", results.time_ms);
        }
    }

    if (fail_count) {
        printf("\nCorrectness: %d tests failed\n", fail_count);
    } else {
        printf("\nCorrectness: All tests passed\n");
    }

    return 0;
}

#include <algorithm>
#include <chrono>
#include <cmath>
#include <cstdint>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <functional>
#include <iostream>
#include <random>
#include <string>
#include <vector>

////////////////////////////////////////////////////////////////////////////////
// Utility Functions

void cuda_check(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << ": "
                  << hipGetErrorString(code) << std::endl;
        exit(1);
    }
}

#define CUDA_CHECK(x) \
    do { \
        cuda_check((x), __FILE__, __LINE__); \
    } while (0)

class GpuMemoryPool {
  public:
    GpuMemoryPool() = default;

    ~GpuMemoryPool();

    GpuMemoryPool(GpuMemoryPool const &) = delete;
    GpuMemoryPool &operator=(GpuMemoryPool const &) = delete;
    GpuMemoryPool(GpuMemoryPool &&) = delete;
    GpuMemoryPool &operator=(GpuMemoryPool &&) = delete;

    void *alloc(size_t size);
    void reset();

  private:
    std::vector<void *> allocations_;
    std::vector<size_t> capacities_;
    size_t next_idx_ = 0;
};

////////////////////////////////////////////////////////////////////////////////
// CPU Reference Implementation (Already Written)

void render_cpu(
    int32_t width,
    int32_t height,
    int32_t n_circle,
    float const *circle_x,
    float const *circle_y,
    float const *circle_radius,
    float const *circle_red,
    float const *circle_green,
    float const *circle_blue,
    float const *circle_alpha,
    float *img_red,
    float *img_green,
    float *img_blue) {

    // Initialize background to white
    for (int32_t pixel_idx = 0; pixel_idx < width * height; pixel_idx++) {
        img_red[pixel_idx] = 1.0f;
        img_green[pixel_idx] = 1.0f;
        img_blue[pixel_idx] = 1.0f;
    }

    // Render circles
    for (int32_t i = 0; i < n_circle; i++) {
        float c_x = circle_x[i];
        float c_y = circle_y[i];
        float c_radius = circle_radius[i];
        for (int32_t y = int32_t(c_y - c_radius); y <= int32_t(c_y + c_radius + 1.0f);
             y++) {
            for (int32_t x = int32_t(c_x - c_radius); x <= int32_t(c_x + c_radius + 1.0f);
                 x++) {
                float dx = x - c_x;
                float dy = y - c_y;
                if (!(0 <= x && x < width && 0 <= y && y < height &&
                      dx * dx + dy * dy < c_radius * c_radius)) {
                    continue;
                }
                int32_t pixel_idx = y * width + x;
                float pixel_red = img_red[pixel_idx];
                float pixel_green = img_green[pixel_idx];
                float pixel_blue = img_blue[pixel_idx];
                float pixel_alpha = circle_alpha[i];
                pixel_red =
                    circle_red[i] * pixel_alpha + pixel_red * (1.0f - pixel_alpha);
                pixel_green =
                    circle_green[i] * pixel_alpha + pixel_green * (1.0f - pixel_alpha);
                pixel_blue =
                    circle_blue[i] * pixel_alpha + pixel_blue * (1.0f - pixel_alpha);
                img_red[pixel_idx] = pixel_red;
                img_green[pixel_idx] = pixel_green;
                img_blue[pixel_idx] = pixel_blue;
            }
        }
    }
}

/// <--- your code here --->

// PSeudo Code

////////////////////////////////////////////////////////////////////////////////
// Optimized GPU Implementation

namespace circles_gpu {

#define THREADS_SCAN (4 * 32)
#define CEIL_DIV(x, y) (((x) + (y) - 1) / (y))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define PAD 32
#define SHMEM_PADDING(idx) ((idx) + ((idx) / PAD))
#define MIN(a, b) ((a) < (b) ? (a) : (b))
#define MAX(a, b) ((a) > (b) ? (a) : (b))

/* TODO: your GPU kernels here... */

__global__ void reduce_uint8(size_t n, uint8_t const *x, uint32_t *out) {

    size_t base = (size_t)blockIdx.x * (size_t)blockDim.x;
    size_t offset = base + (size_t)threadIdx.x;

    extern __shared__ __align__(16) char shmem_raw[];
    uint32_t *shmem = reinterpret_cast<uint32_t *>(shmem_raw);

    if (offset < n) {

        shmem[SHMEM_PADDING(threadIdx.x)] = (uint32_t)(x[offset]);
    }
    //  else {
    //     shmem[SHMEM_PADDING(threadIdx.x)] = 0;
    // }

    __syncthreads();

    for (int i = 1; i < THREADS_SCAN; i <<= 1) {
        uint32_t add = (threadIdx.x >= i) ? shmem[SHMEM_PADDING(threadIdx.x - i)] : 0;
        __syncthreads();
        shmem[SHMEM_PADDING(threadIdx.x)] = add + shmem[SHMEM_PADDING(threadIdx.x)];
        __syncthreads();
    }

    int last = MIN((int)n - (int)(blockIdx.x * blockDim.x), (int)blockDim.x) - 1;
    out[blockIdx.x] = shmem[SHMEM_PADDING(last)];
}

__global__ void reduce(size_t n, uint32_t const *x, uint32_t *out) {

    // uint32_t const *x_block = x + blockIdx.x * blockDim.x;
    size_t base = (size_t)blockIdx.x * (size_t)blockDim.x;
    size_t offset = base + (size_t)threadIdx.x;

    extern __shared__ __align__(16) char shmem_raw[];
    uint32_t *shmem = reinterpret_cast<uint32_t *>(shmem_raw);
    if (offset < n) {
        shmem[SHMEM_PADDING(threadIdx.x)] = x[offset];
    }

    __syncthreads();

    for (int i = 1; i < THREADS_SCAN; i <<= 1) {
        uint32_t add = (threadIdx.x >= i) ? shmem[SHMEM_PADDING(threadIdx.x - i)] : 0;
        __syncthreads();
        shmem[SHMEM_PADDING(threadIdx.x)] = add + shmem[SHMEM_PADDING(threadIdx.x)];
        __syncthreads();
    }

    int last = MIN((int)n - (int)(blockIdx.x * blockDim.x), (int)blockDim.x) - 1;
    out[blockIdx.x] = shmem[SHMEM_PADDING(last)];
}

__global__ void scan_block(size_t n, uint32_t const *x, uint32_t *out) {

    extern __shared__ __align__(16) char shmem_raw[];
    uint32_t *shmem = reinterpret_cast<uint32_t *>(shmem_raw);

    int WARP_SIZE = 32;
    const int tid = threadIdx.x;
    const int lane = tid & 31;
    const int warp = tid >> 5;

    uint32_t val = x[tid];

    for (int i = 1; i < WARP_SIZE; i <<= 1) {
        uint32_t v = __shfl_up_sync(0xFFFFFFFFu, val, i);
        if (lane >= i)
            val += v;
    }

    if (lane == WARP_SIZE - 1) {
        shmem[warp] = val;
    }

    __syncthreads();

    if (warp == 0) {
        uint32_t val2 = shmem[lane];

        for (int i = 1; i < WARP_SIZE; i <<= 1) {
            uint32_t v = __shfl_up_sync(0xffffffff, val2, i);
            if (lane >= i)
                val2 += v;
        }

        shmem[lane] = val2;
    }

    __syncthreads();

    uint32_t to_add = (warp == 0) ? 0 : shmem[warp - 1];
    val += to_add;

    out[tid] = val;
}

__global__ void
scan(size_t n, uint32_t const *x, uint32_t const *end_points, uint32_t *out) {

    extern __shared__ __align__(16) char shmem_raw[];
    uint32_t *shmem = reinterpret_cast<uint32_t *>(shmem_raw);

    size_t base = (size_t)blockIdx.x * (size_t)blockDim.x;
    size_t offset = base + (size_t)threadIdx.x;

    if (offset < n) {
        shmem[SHMEM_PADDING(threadIdx.x)] = x[offset];
    }

    __syncthreads();

    for (int i = 1; i < THREADS_SCAN; i <<= 1) {
        uint32_t add = (threadIdx.x >= i) ? shmem[SHMEM_PADDING(threadIdx.x - i)] : 0;
        __syncthreads();
        shmem[SHMEM_PADDING(threadIdx.x)] = add + shmem[SHMEM_PADDING(threadIdx.x)];
        __syncthreads();
    }
    uint32_t block_carry = (blockIdx.x == 0) ? 0 : end_points[blockIdx.x - 1];
    if (offset < n)
        out[offset] = block_carry + shmem[SHMEM_PADDING(threadIdx.x)];
}

// Returns desired size of scratch buffer in bytes.
size_t get_workspace_size_scan(size_t n) {
    /* TODO: your CPU code here... */
    size_t total = n;
    size_t size = n;
    while (size > THREADS_SCAN) {
        total += size;
        size = CEIL_DIV(size, THREADS_SCAN);
    }

    return 2 * total * sizeof(uint32_t);
}

__global__ void fill_data(uint8_t *data_uint8, uint32_t *data, uint32_t n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n)
        data[idx] = static_cast<uint32_t>(data_uint8[idx]);
}

// 'launch_scan'
//

uint32_t *launch_scan(
    size_t n,
    uint8_t *x,     // pointer to GPU memory
    void *workspace // pointer to GPU memory
    // uint32_t *orig_x // 32 bit representation
) {
    /* TODO: your CPU code here... */
    uint32_t *arr = reinterpret_cast<uint32_t *>(workspace); // size n
    uint8_t *data_uint8 = x;
    uint32_t *data;
    // uint32_t *orig_x;
    // compute sums per block

    size_t size = n;
    size_t offsets[8];
    offsets[0] = 0;
    int iter = 1;

    // first iteration with uint8
    fill_data<<<CEIL_DIV(size, THREADS_SCAN), THREADS_SCAN>>>(data_uint8, arr, size);
    data = arr;
    offsets[0] = size;

    if (size > THREADS_SCAN) {
        size_t blocks = CEIL_DIV(size, THREADS_SCAN);
        // printf("Reducing with %d blocks and %d threads\n", blocks, THREADS_SCAN);
        reduce_uint8<<<blocks, THREADS_SCAN, (THREADS_SCAN + PAD) * sizeof(uint32_t)>>>(
            size,
            data_uint8,
            arr + offsets[iter - 1]);

        size = blocks;
        data = arr + offsets[iter - 1];
        offsets[1] = size + offsets[0];
        iter++;
    }

    while (size > THREADS_SCAN) {
        size_t blocks = CEIL_DIV(size, THREADS_SCAN);
        reduce<<<blocks, THREADS_SCAN, (THREADS_SCAN + PAD) * sizeof(uint32_t)>>>(
            size,
            data,
            arr + offsets[iter - 1]);

        size = blocks;
        data = arr + offsets[iter - 1];

        offsets[iter] = offsets[iter - 1] + size;
        iter++;
    }
    iter--;

    uint32_t *final_block = (iter == 0) ? arr : arr + offsets[iter];
    size_t threads = MIN(THREADS_SCAN, n);

    uint32_t *base_out = (iter == 0) ? arr : final_block + threads;

    scan_block<<<1, threads, (32) * sizeof(uint32_t)>>>(size, final_block, base_out);

    size_t larger_size = threads;
    base_out += larger_size;
    uint32_t *end_points = arr + offsets[iter];
    if (n > THREADS_SCAN) {
        while (iter >= 0) {

            larger_size = (iter == 0) ? n : (offsets[iter] - offsets[iter - 1]);
            size_t blocks = CEIL_DIV(larger_size, THREADS_SCAN);

            uint32_t *data_ptr =
                (iter == 0) ? arr : arr + offsets[iter - 1]; // think this is buggy
            scan<<<blocks, THREADS_SCAN, (THREADS_SCAN + PAD) * sizeof(uint32_t)>>>(
                larger_size,
                data_ptr,
                end_points,
                base_out);

            size = larger_size;
            iter--;
            end_points = base_out;

            base_out += larger_size;
        }
    }
    base_out -= larger_size;

    return base_out;
}

#define THREADS_X 32
#define THREADS_Y 32
#define TILE_SIZE 64

/* TODO: your GPU kernels here... */

__global__ void tile_coverage(
    int32_t width,
    int32_t height,
    int32_t n_circle,
    float const *circle_x,
    float const *circle_y,
    float const *circle_radius,
    uint8_t *circle_map,
    int32_t num_tiles) {

    int32_t circle = blockIdx.x * blockDim.x + threadIdx.x;

    size_t stride = (size_t)n_circle;
    size_t total = (size_t)num_tiles * stride;

    if (circle < n_circle) {

        int tile_width = width / TILE_SIZE;
        int tile_height = height / TILE_SIZE;

        float x = circle_x[circle];
        float y = circle_y[circle];
        float rad = circle_radius[circle];

        int left = (int)MAX((x - rad) / TILE_SIZE, 0);
        int right = (int)MIN((x + rad) / TILE_SIZE, tile_width - 1);
        int top = (int)MAX((y - rad) / TILE_SIZE, 0);
        int bot = (int)MIN((y + rad) / TILE_SIZE, tile_height - 1);

        // go through each corner of the bounding box
        for (int x = left; x < right + 1; x += 1) {
            for (int y = top; y < bot + 1; y += 1) {
                int tile = y * tile_width + x;

                size_t idx = (size_t)tile * stride + (size_t)circle;

                circle_map[idx] = (idx < total);
            }
        }
    }
}
#define MAX_CIRCLES_PER_TILE 44000
__global__ void compact_stream(
    uint32_t n_circle,
    uint32_t *scanned_idxs,
    uint32_t *compacted_idxs
    // uint32_t **ptr_array
    // const float *circle_x,
    // float *x_coal,
    // float **x_coal_ptr
) {

    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n_circle) {
        if (idx == 0) {
            compacted_idxs[0] = 0;
        } else if (scanned_idxs[idx] != scanned_idxs[idx - 1]) {
            uint32_t c_idx = scanned_idxs[idx];

            compacted_idxs[c_idx - 1] = idx;
            // x_coal[c_idx - 1] = circle_x[idx];
        }
    }
    // if (threadIdx.x == 0 && blockIdx.x == 0) {
    //     *ptr_array = compacted_idxs;
    // }
    // *x_coal_ptr = x_coal;
}

#define CIRCLE_STATS 8
__global__ void render_pixels(
    uint32_t *num_circles_per_tile,
    uint32_t num_tiles,
    int32_t width,
    int32_t height,
    int32_t n_circle,
    float const *circle_x,
    float const *circle_y,
    float const *circle_radius,
    float const *circle_red,
    float const *circle_green,
    float const *circle_blue,
    float const *circle_alpha,
    float *img_red,
    float *img_green,
    float *img_blue,
    uint32_t *compacted_stream

    // float **x_coal_ptr
) {

    int blockToTile =
        TILE_SIZE / THREADS_X; // assume threads x and threads y are the same

    int32_t tile_idx = blockIdx.y * gridDim.x + blockIdx.x;

    int32_t base_pixel_row = (blockIdx.y * blockDim.y) * blockToTile + threadIdx.y;
    int32_t base_pixel_col = (blockIdx.x * blockDim.x) * blockToTile + threadIdx.x;

    extern __shared__ __align__(16) char shmem_raw[];
    float *shmem = reinterpret_cast<float *>(shmem_raw);
    const int threads = blockDim.x * blockDim.y;

    // float *x_sh = shmem + threads + 7;
    // float *y_sh = x_sh + threads + 7;
    // float *radius_sh = y_sh + threads + 7;
    // float *alpha_sh = radius_sh + threads + 7;
    // float *red_sh = alpha_sh + threads + 7;
    // float *green_sh = red_sh + threads + 7;
    // float *blue_sh = green_sh + threads + 7;

    uint32_t n_t_circles = num_circles_per_tile[tile_idx];

    // printf("n t  circles: %d\n", n_t_circles);
    uint32_t *tile_circles = compacted_stream + tile_idx * MAX_CIRCLES_PER_TILE;
    // float *x_coal = x_coal_ptr[tile_idx];

    float red_out[4] = {1.0f, 1.0f, 1.0f, 1.0f};
    float green_out[4] = {1.0f, 1.0f, 1.0f, 1.0f};
    float blue_out[4] = {1.0f, 1.0f, 1.0f, 1.0f};

    for (int chunk = 0; chunk < n_t_circles; chunk += threads) {

        int tid = (threadIdx.y * blockDim.x + threadIdx.x);
        int circle_id_id = chunk + tid;

        if (circle_id_id < n_t_circles) {
            uint32_t c_idx = tile_circles[circle_id_id];

            float4 circle_xyra = make_float4(
                circle_x[c_idx],
                circle_y[c_idx],
                circle_radius[c_idx],
                circle_alpha[c_idx]);

            float4 circle_rgbp = make_float4(
                circle_red[c_idx],
                circle_green[c_idx],
                circle_blue[c_idx],
                0.0f // padding (unused)
            );
            reinterpret_cast<float4 *>(&shmem[CIRCLE_STATS * tid])[0] = circle_xyra;
            reinterpret_cast<float4 *>(&shmem[CIRCLE_STATS * tid])[1] = circle_rgbp;

            // x_sh[tid] = circle_x[c_idx];
            // y_sh[tid] = circle_y[c_idx];
            // radius_sh[tid] = circle_radius[c_idx];
            // alpha_sh[tid] = circle_alpha[c_idx];
            // red_sh[tid] = circle_red[c_idx];
            // green_sh[tid] = circle_green[c_idx];
            // blue_sh[tid] = circle_blue[c_idx];
        }

        int loop_iters = threads;
        if (chunk + threads >= n_t_circles) {
            loop_iters = n_t_circles - chunk; // the remainder
        }

        __syncthreads();
        for (int b_i = 0; b_i < blockToTile; ++b_i) {
            for (int b_j = 0; b_j < blockToTile; ++b_j) {

                int b_id = b_i * blockToTile + b_j;
                int32_t pixel_row = base_pixel_row + b_i * THREADS_Y;
                int32_t pixel_col = base_pixel_col + b_j * THREADS_X;

                for (int i = 0; i < loop_iters; ++i) {

                    // uint32_t c_idx = tile_circles[i];
                    float *circle_stats = shmem + CIRCLE_STATS * i;

                    float4 xyra = reinterpret_cast<const float4 *>(circle_stats)[0];
                    float4 rgbp = reinterpret_cast<const float4 *>(circle_stats)[1];

                    float x = xyra.x;
                    float y = xyra.y;
                    float rad = xyra.z;
                    float alpha = xyra.w;

                    float c_red = rgbp.x;
                    float c_green = rgbp.y;
                    float c_blue = rgbp.z;

                    // float x = x_sh[i];
                    // float y = y_sh[i];
                    // float rad = radius_sh[i];
                    // float alpha = alpha_sh[i];
                    // float c_red = red_sh[i];
                    // float c_green = green_sh[i];
                    // float c_blue = blue_sh[i];

                    float dy = pixel_row - y;
                    float dx = pixel_col - x;

                    if (!(dy * dy + dx * dx < rad * rad))
                        continue;

                    red_out[b_id] = c_red * alpha + red_out[b_id] * (1.0f - alpha);
                    green_out[b_id] = c_green * alpha + green_out[b_id] * (1.0f - alpha);
                    blue_out[b_id] = c_blue * alpha + blue_out[b_id] * (1.0f - alpha);
                    // printf("PRINTING\n");
                }
            }
        }
        // __syncthreads();
    }

    for (int i = 0; i < blockToTile; ++i) {
        for (int j = 0; j < blockToTile; ++j) {

            int32_t pixel_row = base_pixel_row + i * THREADS_Y;
            int32_t pixel_col = base_pixel_col + j * THREADS_X;

            int32_t pixel_idx = pixel_row * width + pixel_col;
            int b_id = i * blockToTile + j;

            img_red[pixel_idx] = red_out[b_id];
            img_green[pixel_idx] = green_out[b_id];
            img_blue[pixel_idx] = blue_out[b_id];
        }
    }
}

void launch_render(
    int32_t width,
    int32_t height,
    int32_t n_circle,
    float const *circle_x,      // pointer to GPU memory
    float const *circle_y,      // pointer to GPU memory
    float const *circle_radius, // pointer to GPU memory
    float const *circle_red,    // pointer to GPU memory
    float const *circle_green,  // pointer to GPU memory
    float const *circle_blue,   // pointer to GPU memory
    float const *circle_alpha,  // pointer to GPU memory
    float *img_red,             // pointer to GPU memory
    float *img_green,           // pointer to GPU memory
    float *img_blue,            // pointer to GPU memory
    GpuMemoryPool &memory_pool) {

    // printf("n_circle: %d. size size: %d\n", n_circle, sizeof(size_t));

    // Initialize background to white

    uint32_t threads_c = THREADS_X * THREADS_Y;
    uint32_t blocks_c = CEIL_DIV(n_circle, threads_c);

    // Get circle coverage
    int32_t num_tiles = CEIL_DIV(width, TILE_SIZE) * CEIL_DIV(height, TILE_SIZE);
    size_t circle_map_size = (size_t)num_tiles * (size_t)n_circle * sizeof(uint8_t);

    // printf("cmap size: %lf\n", circle_map_size / (1000 * 1000 * 1000));
    uint8_t *circle_map = reinterpret_cast<uint8_t *>(memory_pool.alloc(circle_map_size));

    tile_coverage<<<blocks_c, threads_c>>>(
        width,
        height,
        n_circle,
        circle_x,
        circle_y,
        circle_radius,
        circle_map,
        num_tiles);
    // return;
    // printf("tile coverage success\n");

    size_t scan_size = get_workspace_size_scan(n_circle);

    uint32_t **tile_circle_idxs =
        reinterpret_cast<uint32_t **>(memory_pool.alloc(num_tiles * sizeof(uint32_t *)));
    float **x_coal_ptr =
        reinterpret_cast<float **>(memory_pool.alloc(num_tiles * sizeof(float *)));
    //[num_tiles]; // pointers to GPU memory
    uint32_t *num_circles_per_tile = reinterpret_cast<uint32_t *>(
        memory_pool.alloc(num_tiles * sizeof(uint32_t))); //[num_tiles];
    void *scan_workspace = memory_pool.alloc(scan_size);

    // size_t mtemp = 0;

    uint32_t *compacted_stream = reinterpret_cast<uint32_t *>(
        memory_pool.alloc(num_tiles * MAX_CIRCLES_PER_TILE * sizeof(uint32_t)));

    for (int i = 0; i < num_tiles; ++i) {

        // REAL
        // printf("scanning %d\n", i);
        uint32_t *scanned_circle_idxs = launch_scan(
            (size_t)n_circle,
            circle_map + (size_t)i * (size_t)n_circle,
            scan_workspace);

        CUDA_CHECK(hipMemcpy(
            num_circles_per_tile + i,
            scanned_circle_idxs + (n_circle - 1),
            sizeof(uint32_t),
            hipMemcpyDeviceToDevice // this must match your pointers
            ));

        compact_stream<<<blocks_c, threads_c>>>(
            n_circle,
            scanned_circle_idxs,
            compacted_stream + i * MAX_CIRCLES_PER_TILE);
    }

    // printf("MAX TEMP: %zu\n", mtemp);

    int tileToBlock = (TILE_SIZE / THREADS_X);
    dim3 threads_p = dim3(THREADS_X, THREADS_Y);
    dim3 blocks_p = dim3(CEIL_DIV(width, TILE_SIZE), CEIL_DIV(height, TILE_SIZE));

    size_t render_shmem = sizeof(float) * (CIRCLE_STATS * THREADS_X * THREADS_Y);
    // CUDA_CHECK(cudaFuncSetAttribute(
    //     render_pixels,
    //     cudaFuncAttributeMaxDynamicSharedMemorySize,
    //     render_shmem));

    render_pixels<<<blocks_p, threads_p, render_shmem>>>(
        num_circles_per_tile,
        num_tiles,
        width,
        height,
        n_circle,
        circle_x,
        circle_y,
        circle_radius,
        circle_red,
        circle_green,
        circle_blue,
        circle_alpha,
        img_red,
        img_green,
        img_blue,
        compacted_stream

        // x_coal_ptr

    );
}

} // namespace circles_gpu

/// <--- /your code here --->

////////////////////////////////////////////////////////////////////////////////
///          YOU DO NOT NEED TO MODIFY THE CODE BELOW HERE.                  ///
////////////////////////////////////////////////////////////////////////////////

GpuMemoryPool::~GpuMemoryPool() {
    for (auto ptr : allocations_) {
        CUDA_CHECK(hipFree(ptr));
    }
}

void *GpuMemoryPool::alloc(size_t size) {
    if (next_idx_ < allocations_.size()) {
        auto idx = next_idx_++;
        if (size > capacities_.at(idx)) {
            CUDA_CHECK(hipFree(allocations_.at(idx)));
            CUDA_CHECK(hipMalloc(&allocations_.at(idx), size));
            CUDA_CHECK(hipMemset(allocations_.at(idx), 0, size));
            capacities_.at(idx) = size;
        }
        return allocations_.at(idx);
    } else {
        void *ptr;
        CUDA_CHECK(hipMalloc(&ptr, size));
        CUDA_CHECK(hipMemset(ptr, 0, size));
        allocations_.push_back(ptr);
        capacities_.push_back(size);
        next_idx_++;
        return ptr;
    }
}

void GpuMemoryPool::reset() {
    next_idx_ = 0;
    for (int32_t i = 0; i < allocations_.size(); i++) {
        CUDA_CHECK(hipMemset(allocations_.at(i), 0, capacities_.at(i)));
    }
}

template <typename Reset, typename F>
double benchmark_ms(double target_time_ms, Reset &&reset, F &&f) {
    double best_time_ms = std::numeric_limits<double>::infinity();
    double elapsed_ms = 0.0;
    while (elapsed_ms < target_time_ms) {
        reset();
        CUDA_CHECK(hipDeviceSynchronize());
        auto start = std::chrono::high_resolution_clock::now();
        f();
        CUDA_CHECK(hipDeviceSynchronize());
        auto end = std::chrono::high_resolution_clock::now();
        double this_ms = std::chrono::duration<double, std::milli>(end - start).count();
        elapsed_ms += this_ms;
        best_time_ms = std::min(best_time_ms, this_ms);
    }
    return best_time_ms;
}

struct Scene {
    int32_t width;
    int32_t height;
    std::vector<float> circle_x;
    std::vector<float> circle_y;
    std::vector<float> circle_radius;
    std::vector<float> circle_red;
    std::vector<float> circle_green;
    std::vector<float> circle_blue;
    std::vector<float> circle_alpha;

    int32_t n_circle() const { return circle_x.size(); }
};

struct Image {
    int32_t width;
    int32_t height;
    std::vector<float> red;
    std::vector<float> green;
    std::vector<float> blue;
};

float max_abs_diff(Image const &a, Image const &b) {
    float max_diff = 0.0f;
    for (int32_t idx = 0; idx < a.width * a.height; idx++) {
        float diff_red = std::abs(a.red.at(idx) - b.red.at(idx));
        float diff_green = std::abs(a.green.at(idx) - b.green.at(idx));
        float diff_blue = std::abs(a.blue.at(idx) - b.blue.at(idx));
        max_diff = std::max(max_diff, diff_red);
        max_diff = std::max(max_diff, diff_green);
        max_diff = std::max(max_diff, diff_blue);
    }
    return max_diff;
}

struct Results {
    bool correct;
    float max_abs_diff;
    Image image_expected;
    Image image_actual;
    double time_ms;
};

enum class Mode {
    TEST,
    BENCHMARK,
};

template <typename T> struct GpuBuf {
    T *data;

    explicit GpuBuf(size_t n) { CUDA_CHECK(hipMalloc(&data, n * sizeof(T))); }

    explicit GpuBuf(std::vector<T> const &host_data) {
        CUDA_CHECK(hipMalloc(&data, host_data.size() * sizeof(T)));
        CUDA_CHECK(hipMemcpy(
            data,
            host_data.data(),
            host_data.size() * sizeof(T),
            hipMemcpyHostToDevice));
    }

    ~GpuBuf() { CUDA_CHECK(hipFree(data)); }
};

Results run_config(Mode mode, Scene const &scene) {
    auto img_expected = Image{
        scene.width,
        scene.height,
        std::vector<float>(scene.height * scene.width, 0.0f),
        std::vector<float>(scene.height * scene.width, 0.0f),
        std::vector<float>(scene.height * scene.width, 0.0f)};

    render_cpu(
        scene.width,
        scene.height,
        scene.n_circle(),
        scene.circle_x.data(),
        scene.circle_y.data(),
        scene.circle_radius.data(),
        scene.circle_red.data(),
        scene.circle_green.data(),
        scene.circle_blue.data(),
        scene.circle_alpha.data(),
        img_expected.red.data(),
        img_expected.green.data(),
        img_expected.blue.data());

    auto circle_x_gpu = GpuBuf<float>(scene.circle_x);
    auto circle_y_gpu = GpuBuf<float>(scene.circle_y);
    auto circle_radius_gpu = GpuBuf<float>(scene.circle_radius);
    auto circle_red_gpu = GpuBuf<float>(scene.circle_red);
    auto circle_green_gpu = GpuBuf<float>(scene.circle_green);
    auto circle_blue_gpu = GpuBuf<float>(scene.circle_blue);
    auto circle_alpha_gpu = GpuBuf<float>(scene.circle_alpha);
    auto img_red_gpu = GpuBuf<float>(scene.height * scene.width);
    auto img_green_gpu = GpuBuf<float>(scene.height * scene.width);
    auto img_blue_gpu = GpuBuf<float>(scene.height * scene.width);

    auto memory_pool = GpuMemoryPool();

    auto reset = [&]() {
        CUDA_CHECK(
            hipMemset(img_red_gpu.data, 0, scene.height * scene.width * sizeof(float)));
        CUDA_CHECK(hipMemset(
            img_green_gpu.data,
            0,
            scene.height * scene.width * sizeof(float)));
        CUDA_CHECK(
            hipMemset(img_blue_gpu.data, 0, scene.height * scene.width * sizeof(float)));
        memory_pool.reset();
    };

    auto f = [&]() {
        circles_gpu::launch_render(
            scene.width,
            scene.height,
            scene.n_circle(),
            circle_x_gpu.data,
            circle_y_gpu.data,
            circle_radius_gpu.data,
            circle_red_gpu.data,
            circle_green_gpu.data,
            circle_blue_gpu.data,
            circle_alpha_gpu.data,
            img_red_gpu.data,
            img_green_gpu.data,
            img_blue_gpu.data,
            memory_pool);
    };

    reset();
    f();

    auto img_actual = Image{
        scene.width,
        scene.height,
        std::vector<float>(scene.height * scene.width, 0.0f),
        std::vector<float>(scene.height * scene.width, 0.0f),
        std::vector<float>(scene.height * scene.width, 0.0f)};

    CUDA_CHECK(hipMemcpy(
        img_actual.red.data(),
        img_red_gpu.data,
        scene.height * scene.width * sizeof(float),
        hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(
        img_actual.green.data(),
        img_green_gpu.data,
        scene.height * scene.width * sizeof(float),
        hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(
        img_actual.blue.data(),
        img_blue_gpu.data,
        scene.height * scene.width * sizeof(float),
        hipMemcpyDeviceToHost));

    float max_diff = max_abs_diff(img_expected, img_actual);

    if (max_diff > 5e-2) {
        return Results{
            false,
            max_diff,
            std::move(img_expected),
            std::move(img_actual),
            0.0,
        };
    }

    if (mode == Mode::TEST) {
        return Results{
            true,
            max_diff,
            std::move(img_expected),
            std::move(img_actual),
            0.0,
        };
    }

    double time_ms = benchmark_ms(1000.0, reset, f);

    return Results{
        true,
        max_diff,
        std::move(img_expected),
        std::move(img_actual),
        time_ms,
    };
}

template <typename Rng>
Scene gen_random(Rng &rng, int32_t width, int32_t height, int32_t n_circle) {
    auto unif_0_1 = std::uniform_real_distribution<float>(0.0f, 1.0f);
    auto z_values = std::vector<float>();
    for (int32_t i = 0; i < n_circle; i++) {
        float z;
        for (;;) {
            z = unif_0_1(rng);
            z = std::max(z, unif_0_1(rng));
            if (z > 0.01) {
                break;
            }
        }
        // float z = std::max(unif_0_1(rng), unif_0_1(rng));
        z_values.push_back(z);
    }
    std::sort(z_values.begin(), z_values.end(), std::greater<float>());

    auto colors = std::vector<uint32_t>{
        0xd32360,
        0xcc9f26,
        0x208020,
        0x2874aa,
    };
    auto color_idx_dist = std::uniform_int_distribution<int>(0, colors.size() - 1);
    auto alpha_dist = std::uniform_real_distribution<float>(0.0f, 0.3f);

    int32_t fog_interval = n_circle / 10;
    float fog_alpha = 0.2;

    auto scene = Scene{width, height};
    float base_radius_scale = 1.0f;
    int32_t i = 0;
    for (float z : z_values) {
        float max_radius = base_radius_scale / z;
        float radius = std::max(1.0f, unif_0_1(rng) * max_radius);
        float x = unif_0_1(rng) * (width + 2 * max_radius) - max_radius;
        float y = unif_0_1(rng) * (height + 2 * max_radius) - max_radius;
        int color_idx = color_idx_dist(rng);
        uint32_t color = colors[color_idx];
        scene.circle_x.push_back(x);
        scene.circle_y.push_back(y);
        scene.circle_radius.push_back(radius);
        scene.circle_red.push_back(float((color >> 16) & 0xff) / 255.0f);
        scene.circle_green.push_back(float((color >> 8) & 0xff) / 255.0f);
        scene.circle_blue.push_back(float(color & 0xff) / 255.0f);
        scene.circle_alpha.push_back(alpha_dist(rng));
        i++;
        if (i % fog_interval == 0 && i + 1 < n_circle) {
            scene.circle_x.push_back(float(width - 1) / 2.0f);
            scene.circle_y.push_back(float(height - 1) / 2.0f);
            scene.circle_radius.push_back(float(std::max(width, height)));
            scene.circle_red.push_back(1.0f);
            scene.circle_green.push_back(1.0f);
            scene.circle_blue.push_back(1.0f);
            scene.circle_alpha.push_back(fog_alpha);
        }
    }

    return scene;
}

constexpr float PI = 3.14159265359f;

Scene gen_overlapping_opaque() {
    int32_t width = 256;
    int32_t height = 256;

    auto scene = Scene{width, height};

    auto colors = std::vector<uint32_t>{
        0xd32360,
        0xcc9f26,
        0x208020,
        0x2874aa,
    };

    int32_t n_circle = 20;
    int32_t n_ring = 4;
    float angle_range = PI;
    for (int32_t ring = 0; ring < n_ring; ring++) {
        float dist = 20.0f * (ring + 1);
        float saturation = float(ring + 1) / n_ring;
        float hue_shift = float(ring) / (n_ring - 1);
        for (int32_t i = 0; i < n_circle; i++) {
            float theta = angle_range * i / (n_circle - 1);
            float x = width / 2.0f - dist * std::cos(theta);
            float y = height / 2.0f - dist * std::sin(theta);
            scene.circle_x.push_back(x);
            scene.circle_y.push_back(y);
            scene.circle_radius.push_back(16.0f);
            auto color = colors[(i + ring * 2) % colors.size()];
            scene.circle_red.push_back(float((color >> 16) & 0xff) / 255.0f);
            scene.circle_green.push_back(float((color >> 8) & 0xff) / 255.0f);
            scene.circle_blue.push_back(float(color & 0xff) / 255.0f);
            scene.circle_alpha.push_back(1.0f);
        }
    }

    return scene;
}

Scene gen_overlapping_transparent() {
    int32_t width = 256;
    int32_t height = 256;

    auto scene = Scene{width, height};

    float offset = 20.0f;
    float radius = 40.0f;
    scene.circle_x = std::vector<float>{
        (width - 1) / 2.0f - offset,
        (width - 1) / 2.0f + offset,
        (width - 1) / 2.0f + offset,
        (width - 1) / 2.0f - offset,
    };
    scene.circle_y = std::vector<float>{
        (height - 1) * 0.75f,
        (height - 1) * 0.75f,
        (height - 1) * 0.25f,
        (height - 1) * 0.25f,
    };
    scene.circle_radius = std::vector<float>{
        radius,
        radius,
        radius,
        radius,
    };
    // 0xd32360
    // 0x2874aa
    scene.circle_red = std::vector<float>{
        float(0xd3) / 255.0f,
        float(0x28) / 255.0f,
        float(0x28) / 255.0f,
        float(0xd3) / 255.0f,
    };
    scene.circle_green = std::vector<float>{
        float(0x23) / 255.0f,
        float(0x74) / 255.0f,
        float(0x74) / 255.0f,
        float(0x23) / 255.0f,
    };
    scene.circle_blue = std::vector<float>{
        float(0x60) / 255.0f,
        float(0xaa) / 255.0f,
        float(0xaa) / 255.0f,
        float(0x60) / 255.0f,
    };
    scene.circle_alpha = std::vector<float>{
        0.75f,
        0.75f,
        0.75f,
        0.75f,
    };
    return scene;
}

Scene gen_simple() {
    /*
        0xd32360,
        0xcc9f26,
        0x208020,
        0x2874aa,
    */
    int32_t width = 256;
    int32_t height = 256;
    auto scene = Scene{width, height};
    scene.circle_x = std::vector<float>{
        (width - 1) * 0.25f,
        (width - 1) * 0.75f,
        (width - 1) * 0.25f,
        (width - 1) * 0.75f,
    };
    scene.circle_y = std::vector<float>{
        (height - 1) * 0.25f,
        (height - 1) * 0.25f,
        (height - 1) * 0.75f,
        (height - 1) * 0.75f,
    };
    scene.circle_radius = std::vector<float>{
        40.0f,
        40.0f,
        40.0f,
        40.0f,
    };
    scene.circle_red = std::vector<float>{
        float(0xd3) / 255.0f,
        float(0xcc) / 255.0f,
        float(0x20) / 255.0f,
        float(0x28) / 255.0f,
    };
    scene.circle_green = std::vector<float>{
        float(0x23) / 255.0f,
        float(0x9f) / 255.0f,
        float(0x80) / 255.0f,
        float(0x74) / 255.0f,
    };
    scene.circle_blue = std::vector<float>{
        float(0x60) / 255.0f,
        float(0x26) / 255.0f,
        float(0x20) / 255.0f,
        float(0xaa) / 255.0f,
    };
    scene.circle_alpha = std::vector<float>{
        1.0f,
        1.0f,
        1.0f,
        1.0f,
    };
    return scene;
}

// Output image writers: BMP file header structure
#pragma pack(push, 1)
struct BMPHeader {
    uint16_t fileType{0x4D42};   // File type, always "BM"
    uint32_t fileSize{0};        // Size of the file in bytes
    uint16_t reserved1{0};       // Always 0
    uint16_t reserved2{0};       // Always 0
    uint32_t dataOffset{54};     // Start position of pixel data
    uint32_t headerSize{40};     // Size of this header (40 bytes)
    int32_t width{0};            // Image width in pixels
    int32_t height{0};           // Image height in pixels
    uint16_t planes{1};          // Number of color planes
    uint16_t bitsPerPixel{24};   // Bits per pixel (24 for RGB)
    uint32_t compression{0};     // Compression method (0 for uncompressed)
    uint32_t imageSize{0};       // Size of raw bitmap data
    int32_t xPixelsPerMeter{0};  // Horizontal resolution
    int32_t yPixelsPerMeter{0};  // Vertical resolution
    uint32_t colorsUsed{0};      // Number of colors in the color palette
    uint32_t importantColors{0}; // Number of important colors
};
#pragma pack(pop)

void write_bmp(
    std::string const &fname,
    uint32_t width,
    uint32_t height,
    const std::vector<uint8_t> &pixels) {
    BMPHeader header;
    header.width = width;
    header.height = height;

    uint32_t rowSize = (width * 3 + 3) & (~3); // Align to 4 bytes
    header.imageSize = rowSize * height;
    header.fileSize = header.dataOffset + header.imageSize;

    std::ofstream file(fname, std::ios::binary);
    file.write(reinterpret_cast<const char *>(&header), sizeof(header));

    // Write pixel data with padding
    std::vector<uint8_t> padding(rowSize - width * 3, 0);
    for (int32_t idx_y = height - 1; idx_y >= 0;
         --idx_y) { // BMP stores pixels from bottom to top
        const uint8_t *row = &pixels[idx_y * width * 3];
        file.write(reinterpret_cast<const char *>(row), width * 3);
        if (!padding.empty()) {
            file.write(reinterpret_cast<const char *>(padding.data()), padding.size());
        }
    }
}

uint8_t float_to_byte(float x) {
    if (x < 0) {
        return 0;
    } else if (x >= 1) {
        return 255;
    } else {
        return x * 255.0f;
    }
}

void write_image(std::string const &fname, Image const &img) {
    auto pixels = std::vector<uint8_t>(img.width * img.height * 3);
    for (int32_t idx = 0; idx < img.width * img.height; idx++) {
        float red = img.red.at(idx);
        float green = img.green.at(idx);
        float blue = img.blue.at(idx);
        // BMP stores pixels in BGR order
        pixels.at(idx * 3) = float_to_byte(blue);
        pixels.at(idx * 3 + 1) = float_to_byte(green);
        pixels.at(idx * 3 + 2) = float_to_byte(red);
    }
    write_bmp(fname, img.width, img.height, pixels);
}

Image compute_img_diff(Image const &a, Image const &b) {
    auto img_diff = Image{
        a.width,
        a.height,
        std::vector<float>(a.height * a.width, 0.0f),
        std::vector<float>(a.height * a.width, 0.0f),
        std::vector<float>(a.height * a.width, 0.0f),
    };
    for (int32_t idx = 0; idx < a.width * a.height; idx++) {
        img_diff.red.at(idx) = std::abs(a.red.at(idx) - b.red.at(idx));
        img_diff.green.at(idx) = std::abs(a.green.at(idx) - b.green.at(idx));
        img_diff.blue.at(idx) = std::abs(a.blue.at(idx) - b.blue.at(idx));
    }
    return img_diff;
}

struct SceneTest {
    std::string name;
    Mode mode;
    Scene scene;
};

int main(int argc, char const *const *argv) {
    auto rng = std::mt19937(0xCA7CAFE);

    auto scenes = std::vector<SceneTest>();
    scenes.push_back({"simple", Mode::TEST, gen_simple()});
    scenes.push_back({"overlapping_opaque", Mode::TEST, gen_overlapping_opaque()});
    scenes.push_back(
        {"overlapping_transparent", Mode::TEST, gen_overlapping_transparent()});
    scenes.push_back(
        {"ten_million_circles",
         Mode::BENCHMARK,
         gen_random(rng, 1024, 1024, 10'000'000)});

    int32_t fail_count = 0;

    int32_t count = 0;
    for (auto const &scene_test : scenes) {
        auto i = count++;
        printf("\nTesting scene '%s'\n", scene_test.name.c_str());
        auto results = run_config(scene_test.mode, scene_test.scene);
        write_image(
            std::string("out/img") + std::to_string(i) + "_" + scene_test.name +
                "_cpu.bmp",
            results.image_expected);
        write_image(
            std::string("out/img") + std::to_string(i) + "_" + scene_test.name +
                "_gpu.bmp",
            results.image_actual);
        if (!results.correct) {
            printf("  Result did not match expected image\n");
            printf("  Max absolute difference: %.2e\n", results.max_abs_diff);
            auto diff = compute_img_diff(results.image_expected, results.image_actual);
            write_image(
                std::string("out/img") + std::to_string(i) + "_" + scene_test.name +
                    "_diff.bmp",
                diff);
            printf(
                "  (Wrote image diff to 'out/img%d_%s_diff.bmp')\n",
                i,
                scene_test.name.c_str());
            fail_count++;
            continue;
        } else {
            printf("  OK\n");
        }
        if (scene_test.mode == Mode::BENCHMARK) {
            printf("  Time: %f ms\n", results.time_ms);
        }
    }

    if (fail_count) {
        printf("\nCorrectness: %d tests failed\n", fail_count);
    } else {
        printf("\nCorrectness: All tests passed\n");
    }

    return 0;
}
